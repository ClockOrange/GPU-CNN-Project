/* Kernel */

#include "cnn.h"

// Free up memory
cnnLayer::~cnnLayer()
{

    hipFree(bias);
    hipFree(weight);
    
	hipFree(output);
	hipFree(prev_value);

	hipFree(backward_output);
	hipFree(backward_prev_value);
	hipFree(backward_weight);
}

cnnLayer::cnnLayer(int Filter, int FilterNum, int ImageSize)
{
	this->Filter = Filter;
	this->FilterNum = FilterNum;
	this->ImageSize = ImageSize;

	output = NULL;
	prev_value = NULL;
	bias   = NULL;
	weight = NULL;

    // allocate memory
    hipMalloc(&output, sizeof(float) * ImageSize);
    hipMalloc(&output, sizeof(float) * FilterNum * Filter);

}

